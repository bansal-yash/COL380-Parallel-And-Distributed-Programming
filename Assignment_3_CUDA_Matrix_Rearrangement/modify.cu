#include "hip/hip_runtime.h"
#include <vector>
#include <utility>
#include <cstring>
#include <hip/hip_runtime.h>
#include "modify.cuh"

using namespace std;

__global__ void fill_freqs(int *cuda_mat, int *cuda_freq_array, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
    {
        int element = cuda_mat[i];
        atomicAdd(&cuda_freq_array[element], 1);
    }
}

__global__ void fill_matrix(int *cuda_ans, int *cuda_prefix_sum, int *cuda_freq_array, int size, int ran)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < ran)
    {
        int freq = cuda_freq_array[i];
        if (freq > 0)
        {
            int start_idx = cuda_prefix_sum[i];
            int end_idx = min(start_idx + freq, size);
            for (int flat_idx = start_idx; flat_idx < end_idx; flat_idx++)
            {
                cuda_ans[flat_idx] = i;
            }
        }
    }
}

void Process_mat(const vector<vector<int>> &matrix, vector<vector<int>> &ans, int ran)
{
    int m = matrix.size();
    int n = matrix[0].size();
    int size = m * n;
    int block_size = 1024;

    hipStream_t stream1;
    hipStreamCreate(&stream1);

    int *cuda_mat, *cuda_freq_array, *cuda_prefix_sum;
    hipMalloc(&cuda_mat, size * sizeof(int));
    hipMalloc(&cuda_freq_array, ran * sizeof(int));
    hipMalloc(&cuda_prefix_sum, ran * sizeof(int));

    hipMemsetAsync(cuda_freq_array, 0, ran * sizeof(int), stream1);
    for (int i = 0; i < m; i++)
    {
        hipMemcpyAsync(&cuda_mat[i * n], matrix[i].data(), n * sizeof(int), hipMemcpyHostToDevice, stream1);
    }

    int num_blocks = (size + block_size - 1) / block_size;
    fill_freqs<<<num_blocks, block_size, 0, stream1>>>(cuda_mat, cuda_freq_array, size);

    hipStreamSynchronize(stream1);

    int *h_freq_array = new int[ran];
    int *h_prefix_sum = new int[ran];

    hipMemcpy(h_freq_array, cuda_freq_array, ran * sizeof(int), hipMemcpyDeviceToHost);

    h_prefix_sum[0] = 0;
    for (int i = 1; i < ran; i++)
    {
        h_prefix_sum[i] = h_prefix_sum[i - 1] + h_freq_array[i - 1];
    }

    hipMemcpy(cuda_prefix_sum, h_prefix_sum, ran * sizeof(int), hipMemcpyHostToDevice);

    delete[] h_freq_array;
    delete[] h_prefix_sum;

    int num_blocks_distribute = (ran + block_size - 1) / block_size;
    fill_matrix<<<num_blocks_distribute, block_size, 0, stream1>>>(cuda_mat, cuda_prefix_sum, cuda_freq_array, size, ran);

    hipStreamSynchronize(stream1);

    for (int i = 0; i < m; i++)
    {
        hipMemcpyAsync(ans[i].data(), &cuda_mat[i * n], n * sizeof(int), hipMemcpyDeviceToHost, stream1);
    }
    hipStreamSynchronize(stream1);

    hipFree(cuda_mat);
    hipFree(cuda_freq_array);
    hipFree(cuda_prefix_sum);
    hipStreamDestroy(stream1);
}

vector<vector<vector<int>>> modify(vector<vector<vector<int>>> &matrices, vector<int> &range)
{
    int num_matrix = matrices.size();
    vector<vector<vector<int>>> res;
    res.reserve(num_matrix);

    for (int a = 0; a < num_matrix; a++)
    {
        const vector<vector<int>> &matrix = matrices[a];
        int m = matrix.size();
        int n = matrix[0].size();
        vector<vector<int>> ans(m, vector<int>(n, 0));
        int ran = range[a] + 1;

        Process_mat(matrix, ans, ran);

        res.emplace_back(std::move(ans));
    }

    return res;
}
